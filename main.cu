#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include<opencv2/video/video.hpp>
#include<opencv2/cudaimgproc.hpp>
#include<opencv2/cudafeatures2d.hpp>
#include<opencv2/cudawarping.hpp>
#include<opencv2/cudacodec.hpp>
#include<opencv2/core/core.hpp>
#include<opencv2/imgproc.hpp>
#include<opencv2/features2d/features2d.hpp>
#include<opencv2/xfeatures2d.hpp>
#include<opencv2/xfeatures2d/cuda.hpp>
#include<opencv2/highgui/highgui.hpp>
#include<opencv2/calib3d/calib3d.hpp>
#include "opencv2/stitching/detail/warpers.hpp"
#include "opencv2/stitching/warpers.hpp"
#include<opencv2/stitching.hpp>
#include <cstdio>
#include<algorithm>
#include<iostream>
#include<cmath>
#include<ctime>
#include<thread>
#include<mutex>
#include<condition_variable>
#include<windows.h>

#include"ransac.h"
#include"ThreadPool.h"
#include"videostitch.h"
#include"utils.h"
using namespace cv;
using namespace std;




//鱼眼摄像头畸变矫正器
class Corrector {
	Mat mapx;
	Mat mapy;
	Mat R;
	Size image_size;
	Mat intrinsic_matrix;
	Mat distortion_coeffs;
	cuda::GpuMat mapx_gpu, mapy_gpu;
public:
	//输入标定数据文件目录，初始化矫正mapx和mapy
	Corrector(string filename) {
		FileStorage fs;
		if (!fs.open(filename, FileStorage::READ)) {
			cout << "打开标定数据失败\n";
			return;
		}
		fs["intrinsic_matrix"] >> intrinsic_matrix;
		fs["distortion_coeffs"] >> distortion_coeffs;
		fs["image_size"] >> image_size;
		fs.release();

		mapx = Mat(image_size, CV_32FC1);
		mapy = Mat(image_size, CV_32FC1);
		R = Mat::eye(3, 3, CV_32F);

		fisheye::initUndistortRectifyMap(intrinsic_matrix, distortion_coeffs, R, intrinsic_matrix, image_size, CV_32FC1, mapx, mapy);
		mapx_gpu.upload(mapx);
		mapy_gpu.upload(mapy);
		cout << "初始化mapx, mapy完成\n";
	}

	//矫正cpu图像
	void FishRemap(Mat &inputArray, Mat &outputArray) {
		cuda::GpuMat src_gpu, img_gpu;
		src_gpu.upload(inputArray);
		cuda::remap(src_gpu, img_gpu, mapx_gpu, mapy_gpu, INTER_LINEAR);
		img_gpu.download(outputArray);
	}

	//矫正gpu图像
	void FishRemap(cuda::GpuMat &inputArray, cuda::GpuMat &outputArray) {
		cuda::remap(inputArray, outputArray, mapx_gpu, mapy_gpu, INTER_LINEAR);
	}
};



//读取视频流线程
void read_thread(Ptr<cudacodec::VideoReader> &cap, cuda::GpuMat &d_frame, mutex &mtx) {
	int flag = 1;
	//int last_t = clock();
	while (flag) {
		unique_lock<mutex> m(mtx);
		//int t1 = clock();
		flag = cap->nextFrame(d_frame);
		//int t2 = clock();
		//cout << "haoshi:" << t2 - t1 << endl;
		m.unlock();
		Sleep(30);
		//int t = clock();
		//cout << "读取每帧：" << t - last_t << endl;
		//last_t = t;
	}
}

//rtsp视频流，输入url初始化
class Video {
	
	string url;
	Ptr<cudacodec::VideoReader> cap;
	cuda::GpuMat d_frame;
	Mat cpu_frame;
	mutex mtx;
public:
	
	Video(string url) :url(url) {
		cap = cudacodec::createVideoReader(url);
	}

	//在子线程中不断读取实时视频
	void run() {
		thread t(read_thread, ref(cap), ref(d_frame), ref(mtx));
		t.detach();
	}

	cuda::GpuMat getGpuMat() {
		return d_frame;
	}

	void show() {	
		unique_lock<mutex> m(mtx);
		d_frame.download(cpu_frame);
		imshow("img", cpu_frame);	
	}

	//获取最新的gpu图像
	void read(cuda::GpuMat &outputArray) {
		unique_lock<mutex> m(mtx);
		outputArray = d_frame;
		m.unlock();
		return;
	}

	//返回互斥锁
	mutex &getMutex() {
		return mtx;
	}
};


mutex mtx;
condition_variable condtion_var;
bool ready_show = 0;
Mat img_show;
int thread_last_time;

//图片拼接流水线线程
void stitch_thread(cuda::GpuMat left_gpu, cuda::GpuMat right_gpu, int img_time, VideoStitcher &stitcher) {
	Mat res;
	if (stitcher.stitch(left_gpu, right_gpu, res) != 0) {
		cout << "拼接失败\n";
		return;
	}
	int now_time = clock();
	if (now_time < thread_last_time)	return;
	unique_lock<mutex> lck(mtx);
	thread_last_time = img_time;
	res.copyTo(img_show);
	ready_show = 1;	
	condtion_var.notify_one();
	lck.unlock();
	return;
}

//显示结果图片线程
void show_thread(int frame_time) {
	int sum = 0;
	int cnt = 0;
	int last_t = clock();
	while (1) {
		unique_lock<mutex> lck(mtx);
		while (ready_show == 0)	condtion_var.wait(lck);
		ready_show = 0;
		int t = clock();
		if (t - last_t < frame_time)	Sleep(frame_time - (t - last_t));
		//resize(img_show, img_show, Size(0, 0), 0.5, 0.5, INTER_LINEAR);
		imshow("res", img_show);
		waitKey(1);	
		t = clock();
		sum += t - last_t;
		cnt++;
		cout << "流水线******最终单帧耗时:" << t - last_t << " ,平均每帧耗时:" << sum / cnt << endl;
		last_t = t;
		
		//Sleep(10);
	}
	
}



int main()
{
	string url_left = "rtsp://192.168.31.12:554/user=admin&password=&channel=1&stream=0.sdp?real_stream";
	string url_right = "rtsp://192.168.31.24:554/user=admin&password=&channel=1&stream=0.sdp?real_stream";
	string filename = "intrinsics.xml";
	Corrector corrector(filename);
	int opt = 4;

	//rtsp视频
	if (opt == 1) {
		Video cap_left(url_left);
		Video cap_right(url_right);
		cap_left.run();
		cap_right.run();
		namedWindow("left");
		namedWindow("right");

		cuda::GpuMat left_gpu, right_gpu;
		while (1) {
			cap_left.read(left_gpu);
			cap_right.read(right_gpu);
			corrector.FishRemap(left_gpu, left_gpu);
			corrector.FishRemap(right_gpu, right_gpu);
			cuda::resize(left_gpu, left_gpu, Size(0, 0), 0.5, 0.5);
			cuda::resize(right_gpu, right_gpu, Size(0, 0), 0.5, 0.5);
			Mat left, right;
			left_gpu.download(left);
			right_gpu.download(right);
			imshow("left", left);
			imshow("right", right);
			if (waitKey(40) == 13)	break;
		}
		destroyAllWindows();

		cap_left.read(left_gpu);
		cap_right.read(right_gpu);
		corrector.FishRemap(left_gpu, left_gpu);
		corrector.FishRemap(right_gpu, right_gpu);
		VideoStitcher stitcher;
		stitcher.init(left_gpu, right_gpu);

		//显示结果线程
		thread showThread(show_thread, 30);
		ThreadPool pool(16);

		while (1) {
			cap_left.read(left_gpu);
			cap_right.read(right_gpu);

			//cuda::cvtColor(src_left_gpu, src_left_gpu, COLOR_BGRA2BGR);
			//cuda::cvtColor(src_right_gpu, src_right_gpu, COLOR_BGRA2BGR);
			//cout << "width:" << src_left_gpu.size() << endl;

			corrector.FishRemap(left_gpu, left_gpu);
			corrector.FishRemap(right_gpu, right_gpu);

			//cuda::resize(src_left_gpu, src_left_gpu, Size(0, 0), 0.5, 0.5);
			//cuda::resize(src_right_gpu, src_right_gpu, Size(0, 0), 0.5, 0.5);

			//Mat src_left, src_right;
			//src_left_gpu.download(src_left);
			//src_right_gpu.download(src_right);
			//imshow("left", src_left);
			//imshow("right", src_right);
			//imwrite("day left.jpg", src_left);
			//imwrite("day right.jpg", src_right);
			//waitKey(1);
			int now = clock();
			pool.enqueue(stitch_thread, left_gpu, right_gpu, now, stitcher);
			Sleep(10);
		}

		showThread.join();
	}
	//靖世九柱，流水线
	if (opt == 2) {
		int last_t = clock();
		thread_last_time = clock();
		

		Mat left = imread("left.jpg");
		Mat right = imread("right.jpg");
		resize(left, left, Size(0, 0), 0.5, 0.5, INTER_LINEAR);
		resize(right, right, Size(0, 0), 0.5, 0.5, INTER_LINEAR);
		cvtColor(left, left, COLOR_BGR2BGRA);
		cvtColor(right, right, COLOR_BGR2BGRA);
		cuda::GpuMat left_gpu, right_gpu;
		left_gpu.upload(left);
		right_gpu.upload(right);
	
		VideoStitcher stitcher;
		stitcher.init(left_gpu, right_gpu);

		thread showThread(show_thread, 20);
		ThreadPool pool(16);
		while (1) {
			Sleep(30);
			int now = clock();
			cuda::GpuMat l = left_gpu.clone();
			cuda::GpuMat r = right_gpu.clone();
			pool.enqueue(stitch_thread, l, r, now, stitcher);
		}
		
		showThread.join();
	}
	//读入实验室视频，流水线
	else if (opt == 3) {
		Ptr<cudacodec::VideoReader> cap_left = cudacodec::createVideoReader(string("day left.mp4"));
		Ptr<cudacodec::VideoReader> cap_right = cudacodec::createVideoReader(string("day right.mp4"));
		cuda::GpuMat left_gpu, right_gpu;

		if (!cap_left->nextFrame(left_gpu))		return -1;
		if (!cap_right->nextFrame(right_gpu))	return -1;
		corrector.FishRemap(left_gpu, left_gpu);
		corrector.FishRemap(right_gpu, right_gpu);
		VideoStitcher stitcher;
		stitcher.init(left_gpu, right_gpu);

		//显示结果线程
		thread showThread(show_thread, 10);
		ThreadPool pool(16);

		int last_t = clock();
		int tt = 1;
		while (1) {
			if (!cap_left->nextFrame(left_gpu))		break;
			if (!cap_right->nextFrame(right_gpu))	break;
			
			//cuda::cvtColor(src_left_gpu, src_left_gpu, COLOR_BGRA2BGR);
			//cuda::cvtColor(src_right_gpu, src_right_gpu, COLOR_BGRA2BGR);
			//cout << "width:" << src_left_gpu.size() << endl;

			corrector.FishRemap(left_gpu, left_gpu);
			corrector.FishRemap(right_gpu, right_gpu);

			//cuda::resize(src_left_gpu, src_left_gpu, Size(0, 0), 0.5, 0.5);
			//cuda::resize(src_right_gpu, src_right_gpu, Size(0, 0), 0.5, 0.5);

			//Mat src_left, src_right;
			//src_left_gpu.download(src_left);
			//src_right_gpu.download(src_right);
			//imshow("left", src_left);
			//imshow("right", src_right);
			//imwrite("day left.jpg", src_left);
			//imwrite("day right.jpg", src_right);
			//waitKey(1);
			int now = clock();
			pool.enqueue(stitch_thread, left_gpu, right_gpu, now, stitcher);
			Sleep(10);
		}

		showThread.join();
	}
	//固定摄像头，读入实验室视频
	else if (opt == 4){
		VideoStitcher video_stitcher;
		Ptr<cudacodec::VideoReader> cap_left = cudacodec::createVideoReader(string("day left.mp4"));
		Ptr<cudacodec::VideoReader> cap_right = cudacodec::createVideoReader(string("day right.mp4"));
		cuda::GpuMat left_gpu, right_gpu;

		if (!cap_left->nextFrame(left_gpu))		return -1;
		if (!cap_right->nextFrame(right_gpu))	return -1;	
		Mat temp;
		left_gpu.download(temp);
		imwrite("实验室.jpg", temp);
		return 0;
		corrector.FishRemap(left_gpu, left_gpu);
		corrector.FishRemap(right_gpu, right_gpu);
		//cuda::resize(left_gpu, left_gpu, Size(0, 0), 0.6, 0.6);
		//cuda::resize(right_gpu, right_gpu, Size(0, 0), 0.6, 0.6);
		cout << "left size:" << left_gpu.size() << endl;
		cout << "right size:" << right_gpu.size() << endl;
		int t1 = clock();
		if (video_stitcher.init(left_gpu, right_gpu) != 0)	return -1;
		int t2 = clock();
		cout << "初始化耗时: " << t2 - t1 << endl;
		cout << "初始化成功\n";

		


		int last_t = clock();
		while (1) {
			if (!cap_left->nextFrame(left_gpu))		break;
			if (!cap_right->nextFrame(right_gpu))	break;

			int t3 = clock();
			corrector.FishRemap(left_gpu, left_gpu);
			corrector.FishRemap(right_gpu, right_gpu);
			int t4 = clock();
			cout << "畸变矫正:" << t4 - t3 << endl;

			//cuda::resize(left_gpu, left_gpu, Size(0, 0), 0.6, 0.6);
			//cuda::resize(right_gpu, right_gpu, Size(0, 0), 0.6, 0.6);

			Mat result;
			if (video_stitcher.stitch(left_gpu, right_gpu, result) != 0) {
				cout << "拼接失败\n";
				continue;
			}

			//resize(result, result, Size(0, 0), 0.3, 0.3);
			int t5 = clock();
			imshow("res", result);
			waitKey(1);
			int t6 = clock();
			cout << "显示图像:" << t6 - t5 << endl;

			int t = clock();
			cout << "每帧耗时:" << t - last_t << endl;
			last_t = t;
			//waitKey(0);
		}
	}
	//靖世九柱，固定摄像头
	else if (opt == 5){
		Mat left = imread("left.jpg");
		Mat right = imread("right.jpg");
		resize(left, left, Size(0, 0), 0.5, 0.5, INTER_LINEAR);
		resize(right, right, Size(0, 0), 0.5, 0.5, INTER_LINEAR);
		cout << "left size:" << left.size() << endl;
		cout << "right size:" << right.size() << endl;
		cuda::GpuMat left_gpu, right_gpu;
		left_gpu.upload(left);
		right_gpu.upload(right);
		cuda::cvtColor(left_gpu, left_gpu, COLOR_BGR2BGRA);
		cuda::cvtColor(right_gpu, right_gpu, COLOR_BGR2BGRA);

		VideoStitcher video_stitcher;
		if (video_stitcher.init(left_gpu, right_gpu) != 0)	return -1;
		cout << "初始化成功\n";

		int last_t = clock();
		while (1) {
			Mat result;
			if (video_stitcher.stitch(left_gpu, right_gpu, result) != 0) {
				cout << "拼接失败\n";
				continue;
			}

			//resize(result, result, Size(0, 0), 0.6, 0.6);
			imshow("res", result);
			waitKey(0);
			int t = clock();
			cout << "每帧耗时:" << t - last_t << endl;
			last_t = t;
		}
	}
	//测试
	else if (opt == 6) {
		VideoCapture cap2;
		cap2.open("dayleft.avi");
		Mat a;
		if (cap2.isOpened())	cout << "打开成功\n";
		while (1) {
			int t1 = clock();
			cap2.read(a);
			int t2 = clock();
			cout <<"a\n" << a;
			cout << "耗时 " << t2 - t1 << endl;
			imshow("a", a);
			waitKey(10);
		}
	}

}

// Helper function for using CUDA to add vectors in parallel.

